#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

#include <iostream>

// Simple CUDA kernel
__global__ void simpleKernel(int *d_data) {
  int idx = threadIdx.x;
  d_data[idx] = idx;
}

class CudaTest : public ::testing::Test {
 protected:
  int *d_data = nullptr;
  int *h_data = nullptr;
  const int dataSize = 10;

  // Set up before each test
  void SetUp() override {
    // Allocate host and device memory
    h_data = new int[dataSize];
    ASSERT_EQ(hipMalloc(&d_data, dataSize * sizeof(int)), hipSuccess)
        << "Failed to allocate device memory";
  }

  // Tear down after each test
  void TearDown() override {
    // Free device and host memory
    hipFree(d_data);
    delete[] h_data;
  }
};

// Test if CUDA kernel runs correctly and produces the expected results
TEST_F(CudaTest, SimpleKernelTest) {
  // Launch kernel with 1 block of dataSize threads
  simpleKernel<<<1, dataSize>>>(d_data);

  // Copy the results back to the host
  ASSERT_EQ(hipMemcpy(
                h_data, d_data, dataSize * sizeof(int), hipMemcpyDeviceToHost),
            hipSuccess)
      << "Failed to copy data from device to host";

  // Verify the result
  for (int i = 0; i < dataSize; i++) {
    EXPECT_EQ(h_data[i], i) << "Kernel output mismatch at index " << i;
  }

  // Check for any errors in kernel execution
  ASSERT_EQ(hipGetLastError(), hipSuccess) << "CUDA kernel execution failed";
}

int main(int argc, char **argv) {
  testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}